
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 20
#define BLOCK_DIM 20
double mat_a[N][N]; //matriz A
double mat_b[N][N]; //matrz B
double mat_result[N][N]; //matriz C

//Contadores de los loops for
int i,j,m;

//Flag para imprimir los resultados
int flag;

__global__ void suma(double *A, double *B, double *C){

	//índices de los hilos
	int columna = blockIdx.x * blockDim.x + threadIdx.x;
	int renglon = blockIdx.y * blockDim.y + threadIdx.y;
	int indice = columna + renglon*N;

	//suma
	if(columna < N && renglon < N){
		C[indice] = A[indice] + B[indice];
	}
}

void inicializa_matrices();
void imprime_matrices();

int main(int argc, char *argv[]){

	//Inicializa matrices A y B
	inicializa_matrices();

	//Se imprimen resultados?
	flag = atoi(argv[1]);


//Variables utilizadas por el device
int size = N*N*sizeof(double);
double *pA, *pB, *pC;

//Memory allocation en el device
hipMalloc((void**)&pA, size);
hipMalloc((void**)&pB, size);
hipMalloc((void**)&pC, size);

//Se copian las matrices del host al device
hipMemcpy(pA, mat_a, size, hipMemcpyHostToDevice);
hipMemcpy(pB, mat_b, size, hipMemcpyHostToDevice);


dim3 dimBlock(BLOCK_DIM,BLOCK_DIM);
dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));
suma<<<dimGrid,dimBlock>>>(pA,pB,pC);

hipMemcpy(mat_result, pC, size, hipMemcpyDeviceToHost);

	if (flag !=0){
		imprime_matrices();	
	}

hipFree(pA); 
hipFree(pB); 
hipFree(pC);

return 0;
}

void inicializa_matrices()
{
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            mat_a[i][j] = i + j;
        }
    }
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            mat_b[i][j] = i*j;
        }
    }
}
void imprime_matrices()
{   printf("Matriz A \n");
    for (i = 0; i < N; i++) {
        printf("\n");
        for (j = 0; j < N; j++)
            printf("%8.2f  ", mat_a[i][j]);
    }
    printf("\n\n\n");
    printf("Matriz B \n");
    for (i = 0; i < N; i++) {
        printf("\n");
        for (j = 0; j < N; j++)
            printf("%8.2f  ", mat_b[i][j]);
    }
    printf("\n\n\n");
    printf("Matriz C = A + B\n");
    for (i = 0; i < N; i++) {
        printf("\n");
        for (j = 0; j < N; j++)
            printf("%8.2f  ", mat_result[i][j]);
    }
    printf("\n\n");
}
