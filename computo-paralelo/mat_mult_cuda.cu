
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 20
#define BLOCK_DIM 20
double mat_a[N][N]; //matriz A
double mat_b[N][N]; //matrz B
double mat_result[N][N]; //matriz C

//Contadores de los loops for
int i,j,m;

//Flag para imprimir los resultados
int flag;

__global__ void multiplica(double *A, double *B, double *C, int dim) {

	//índices de los hilos
	int columna = threadIdx.x + blockDim.x * blockIdx.x;
	int renglon = threadIdx.y + blockDim.y * blockIdx.y;

	//multiplicación
	int k;
	double suma = 0;
	if(columna < dim && renglon < dim){

		for(k = 0; k< dim; k++)
			suma = suma + A[renglon*dim + k]*B[k*dim + columna];
		C[renglon*dim + columna] = suma;
	}
}

void inicializa_matrices();
void imprime_matrices();

int main(int argc, char *argv[]){

	//Inicializa matrices A y B
	inicializa_matrices();

	//Se imprimen resultados?
	flag = atoi(argv[1]);


//Variables utilizadas por el device
int size = N*N*sizeof(double);
double *pA, *pB, *pC;

//Memory allocation en el device
hipMalloc((void**)&pA, size);
hipMalloc((void**)&pB, size);
hipMalloc((void**)&pC, size);

//Se copian las matrices del host al device
hipMemcpy(pA, mat_a, size, hipMemcpyHostToDevice);
hipMemcpy(pB, mat_b, size, hipMemcpyHostToDevice);


dim3 dimBlock(N,N);
dim3 dimGrid(1,1);
multiplica<<<dimGrid,dimBlock>>>(pA,pB,pC,N);

hipMemcpy(mat_result, pC, size, hipMemcpyDeviceToHost);

	if (flag !=0){
		imprime_matrices();	
	}

hipFree(pA); 
hipFree(pB); 
hipFree(pC);

return 0;
}

void inicializa_matrices()
{
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            mat_a[i][j] = i + j;
        }
    }
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            mat_b[i][j] = i*j;
        }
    }
}
void imprime_matrices()
{   printf("Matriz A \n");
    for (i = 0; i < N; i++) {
        printf("\n");
        for (j = 0; j < N; j++)
            printf("%8.2f  ", mat_a[i][j]);
    }
    printf("\n\n\n");
    printf("Matriz B \n");
    for (i = 0; i < N; i++) {
        printf("\n");
        for (j = 0; j < N; j++)
            printf("%8.2f  ", mat_b[i][j]);
    }
    printf("\n\n\n");
    printf("Matriz C = A * B\n");
    for (i = 0; i < N; i++) {
        printf("\n");
        for (j = 0; j < N; j++)
            printf("%8.2f  ", mat_result[i][j]);
    }
    printf("\n\n");
}
